﻿#include <hipfft/hipfft.h>

#include "kernel.cuh"



bool cuda_fft(const std::vector<float>& real_in, std::vector<std::complex<float>>** cpx_out, defs::RfDataDims dims)
{

	hipfftHandle fwd_plan, inv_plan;
	hipStream_t stream = nullptr;

	uint fft_size = dims.sample_count;
	uint batch_count = dims.element_count * dims.tx_count;

	uint unit_count = fft_size * batch_count;

	float* d_input = nullptr;
	hipfftComplex* d_output = nullptr;

	// Because the input is real the right half of the fft isn't calculated
	uint output_size = (fft_size / 2 + 1) * batch_count;

	// Make the output vector the size of the full signal so we can hilbert transform it
	*cpx_out = new std::vector<std::complex<float>>((fft_size) * batch_count);
	

	RETURN_IF_ERROR(hipfftCreate(&fwd_plan), "Failed to create forward plan.\n")
	RETURN_IF_ERROR(hipfftCreate(&inv_plan), "Failed to create inverse plan.\n")

	RETURN_IF_ERROR(hipfftPlan1d(&fwd_plan, fft_size, HIPFFT_R2C, batch_count), "Failed to configure forward plan.\n")
	RETURN_IF_ERROR(hipfftPlan1d(&inv_plan, fft_size, HIPFFT_C2R, batch_count), "Failed to configure inverse plan.\n")

//	RETURN_IF_ERROR(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking), "Failed to create stream.\n")

//	RETURN_IF_ERROR(hipfftSetStream(fwd_plan, stream), "Failed to set forward stream.\n")
//	RETURN_IF_ERROR(hipfftSetStream(inv_plan, stream), "Failed to set inverse stream.\n")

	RETURN_IF_ERROR(hipMalloc((void**)&d_input, sizeof(float) * real_in.size()), "Failed to malloc input array")
	RETURN_IF_ERROR(hipMalloc((void**)&d_output, sizeof(hipfftComplex) * real_in.size()), "Failed to malloc output array")

	RETURN_IF_ERROR(hipMemcpy(d_input, real_in.data(), sizeof(float) * real_in.size(), hipMemcpyHostToDevice), "Failed to memcpy input.\n")


	hipfftResult_t result = hipfftExecR2C(fwd_plan, d_input, d_output);

	RETURN_IF_ERROR(result, "Forward fft failed.")


	RETURN_IF_ERROR(hipMemcpy((*cpx_out)->data(), d_output, sizeof(std::complex<float>)* output_size, hipMemcpyDeviceToHost), "Failed to copy forward result from host.\n")

	//RETURN_IF_ERROR(hipStreamSynchronize(stream), "Failed to sync stream.\n")

	return true;
}






