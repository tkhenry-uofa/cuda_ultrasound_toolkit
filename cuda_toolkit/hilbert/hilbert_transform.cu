﻿#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>

#include <chrono>

#include "hilbert_transform.cuh"

__host__ bool
hilbert::plan_hilbert(int sample_count, int channel_count)
{
	hipfftResult_t fft_result;

	// The FFT fails with CUDA_INTERNAL_ERROR if we don't estimate first, this only happens when FFT size isnt a power of 2
	// No idea what the cause it, it isn't in the docs anywhere.
	size_t work_size = 0;
	int dimensions[] = { sample_count };

	fft_result = hipfftEstimateMany(1, &sample_count, dimensions, 1, sample_count, dimensions, 1, sample_count, HIPFFT_R2C, channel_count, &work_size);
	FFT_RETURN_IF_ERROR(fft_result, "Failed to estimate forward plan.")

	fft_result = hipfftPlanMany(&(Session.forward_plan), 1, &sample_count, dimensions, 1, sample_count, dimensions, 1, sample_count, HIPFFT_R2C, channel_count);
	FFT_RETURN_IF_ERROR(fft_result, "Failed to create forward plan.")

	fft_result = hipfftPlanMany(&(Session.inverse_plan), 1, &sample_count, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, channel_count);
	FFT_RETURN_IF_ERROR(fft_result, "Failed to configure inverse plan.")

	return true;
}

__host__ bool 
hilbert::hilbert_transform(float* d_input, hipfftComplex* d_output)
{
	hipfftResult_t fft_result;

	
	size_t output_size = Session.decoded_dims.x * Session.decoded_dims.y * Session.decoded_dims.z * sizeof(float);
	CUDA_THROW_IF_ERROR(hipMemset(d_output, 0x00, output_size));

	// Exec forward transform
	fft_result = hipfftExecR2C(Session.forward_plan, d_input, d_output);
	FFT_RETURN_IF_ERROR(fft_result, "Failed to execute forward plan.")

	// Exec reverse transform
	fft_result = hipfftExecC2C(Session.inverse_plan, d_output, d_output, HIPFFT_BACKWARD);
	FFT_RETURN_IF_ERROR(fft_result, "Failed to execute inverse plan.")

	return true;
}
