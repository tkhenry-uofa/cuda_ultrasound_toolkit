#include "hip/hip_runtime.h"
﻿#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>

#include <chrono>

#include "hilbert_transform.cuh"

__device__ inline float2 complex_multiply_f2(float2 a, float2 b) {
	return make_float2(a.x * b.x - a.y * b.y,
		a.x * b.y + a.y * b.x);
}


__global__ void
hilbert::kernels::filter(hipComplex* data, uint sample_count, uint cutoff_sample)
{
	uint sample_idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (sample_idx < sample_count && sample_idx > cutoff_sample)
	{
		data[blockIdx.y * sample_count + sample_idx] = { 0.0f, 0.0f };
	}
	else if (sample_idx == 0 || sample_idx == cutoff_sample)
	{
		hipComplex value = data[blockIdx.y * sample_count + sample_idx];
		data[blockIdx.y * sample_count + sample_idx] = SCALE_F2(value, 0.5f);
	}
}

__global__ void
hilbert::kernels::scale_and_filter(hipComplex* spectrums, hipComplex* filter_kernel, uint sample_count)
{
	uint sample_idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (sample_idx > (sample_count >> 1)) return; // We only need to process the first half of the spectrum.

	float scale_factor = 1.0f / (float)sample_count;
	// Scale the DC and Nyquist components by 0.5 compared to the rest of the spectrum (analytic signal)
	if (sample_idx == 0 || sample_idx == (sample_count >> 1)) scale_factor *= 0.5f; 

	uint channel_offset = blockIdx.y * sample_count;
	spectrums[channel_offset + sample_idx] = SCALE_F2(spectrums[channel_offset + sample_idx], scale_factor);

	if (!filter_kernel) return; // No filter kernel, just scale the spectrum

	spectrums[channel_offset + sample_idx] =
		complex_multiply_f2(spectrums[channel_offset + sample_idx], filter_kernel[sample_idx]);

}

__host__ bool
hilbert::setup_filter(int signal_length, int filter_length, const float* filter)
{
	size_t final_size = signal_length * sizeof(hipComplex);
	hipComplex* d_filter;
	CUDA_RETURN_IF_ERROR(hipMalloc((void**)&d_filter, final_size));
	CUDA_RETURN_IF_ERROR(hipMemset(d_filter, 0x00, final_size)); // Padded with zeros to signal_length
	CUDA_FLOAT_TO_COMPLEX(filter, d_filter, filter_length);

	hipfftHandle plan;
	CUFFT_RETURN_IF_ERR(hipfftPlan1d(&plan, signal_length, HIPFFT_C2C, 1));
	CUFFT_RETURN_IF_ERR(hipfftExecC2C(plan, d_filter, d_filter, HIPFFT_BACKWARD));
	Session.d_match_filter = d_filter;
	hipfftDestroy(plan);

	return true;
}

__host__ bool
hilbert::f_domain_filter(hipComplex* input)
{
	uint sample_count = Session.decoded_dims.x;
	uint cutoff_sample = (uint)floor((float)sample_count / 2.0f) + 1; // We know the second half of each spectrum is all 0's

	uint channel_count = Session.decoded_dims.y * Session.decoded_dims.z;
	uint grid_length = (uint)ceil((double)cutoff_sample / MAX_THREADS_PER_BLOCK);

	dim3 grid_dims = { grid_length, channel_count, 1 };
	dim3 block_dims = { MAX_THREADS_PER_BLOCK, 1, 1 };

	kernels::scale_and_filter << <grid_dims, block_dims >> > (input, Session.d_match_filter, sample_count);
	CUDA_RETURN_IF_ERROR(hipGetLastError());
	CUDA_RETURN_IF_ERROR(hipDeviceSynchronize());

	return true;
}

__host__ bool
hilbert::plan_hilbert(int sample_count, int channel_count)
{
	// The FFT fails with CUDA_INTERNAL_ERROR if we don't estimate first, this only happens when FFT size isnt a power of 2
	// No idea what the cause it, it isn't in the docs anywhere.
	size_t work_size = 0;

	int data_length = sample_count * channel_count;
	int double_l = data_length * 2;

	//CUFFT_THROW_IF_ERR(hipfftEstimateMany(1, &sample_count, &sample_count, 1, sample_count, &sample_count, 1, sample_count, HIPFFT_R2C, channel_count, &work_size));

	CUFFT_RETURN_IF_ERR(hipfftPlanMany(&(Session.forward_plan), 1, &sample_count, &data_length, 1, sample_count, &data_length, 1, sample_count, HIPFFT_R2C, channel_count));
	CUFFT_RETURN_IF_ERR(hipfftPlanMany(&(Session.inverse_plan), 1, &sample_count, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, channel_count));

	CUFFT_RETURN_IF_ERR(hipfftPlanMany(&(Session.strided_plan), 1, &sample_count, &data_length, 2, sample_count * 2, &data_length, 1, sample_count, HIPFFT_R2C, channel_count));
	return true;
}

__host__ bool 
hilbert::hilbert_transform(float* d_input, hipComplex* d_output)
{
	size_t output_size = Session.decoded_dims.x * Session.decoded_dims.y * Session.decoded_dims.z * sizeof(hipComplex);

	CUDA_RETURN_IF_ERROR(hipMemset(d_output, 0x00, output_size));

	CUFFT_RETURN_IF_ERR(hipfftExecR2C(Session.forward_plan, d_input, d_output));

	CUDA_RETURN_IF_ERROR(hipGetLastError());
	CUDA_RETURN_IF_ERROR(hipDeviceSynchronize());
	hilbert::f_domain_filter(d_output);
	CUFFT_RETURN_IF_ERR(hipfftExecC2C(Session.inverse_plan, d_output, d_output, HIPFFT_BACKWARD));

	CUDA_RETURN_IF_ERROR(hipGetLastError());
	CUDA_RETURN_IF_ERROR(hipDeviceSynchronize());
	return true;
}

__host__ bool
hilbert::hilbert_transform_strided(float* d_input, hipComplex* d_output)
{
	size_t output_size = (size_t)Session.decoded_dims.x * Session.decoded_dims.y * Session.decoded_dims.z * sizeof(hipComplex);

	CUDA_RETURN_IF_ERROR(hipMemset(d_output, 0x00, output_size));

	CUFFT_RETURN_IF_ERR(hipfftExecR2C(Session.strided_plan, d_input, d_output));

	float scale = 1 / ((float)Session.decoded_dims.x / 2);

	float* sample = (float*)d_output;
	///std::cout << "First input value: Re:" << sample_value(d_input+ Session.decoded_dims.x) << " Im: " << sample_value(d_input+1+ Session.decoded_dims.x) << std::endl;
	//std::cout << "First output value: Re:" << sample_value(sample + Session.decoded_dims.x) * scale << " Im: " << sample_value(sample + 1 + Session.decoded_dims.x) * scale << std::endl;
	//hilbert::f_domain_filter(d_output, Session.decoded_dims.x/2);
	//std::cout << "First output value: Re:" << sample_value(sample + Session.decoded_dims.x) * scale << " Im: " << sample_value(sample + 1 + Session.decoded_dims.x) * scale << std::endl;
	//hilbert::f_domain_filter(d_output, 1350);
	CUFFT_RETURN_IF_ERR(hipfftExecC2C(Session.inverse_plan, d_output, d_output, HIPFFT_BACKWARD));
	return true;
}


