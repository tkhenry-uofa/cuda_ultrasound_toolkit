﻿#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>

#include <chrono>

#include "hilbert_transform.cuh"

__host__ bool
hilbert::plan_hilbert(int sample_count, int channel_count, hipfftHandle* fwd_handle, hipfftHandle* inv_handle)
{
	hipfftResult_t fft_result;

	fft_result = hipfftCreate(fwd_handle);
	FFT_RETURN_IF_ERROR(fft_result, "Failed to create forward plan.\n")
	fft_result = hipfftCreate(fwd_handle);
	FFT_RETURN_IF_ERROR(fft_result, "Failed to create inverse plan.\n")

	// The FFT fails with CUDA_INTERNAL_ERROR if we don't estimate first, this only happens when FFT size isnt a power of 2
	// No idea what the cause it, it isn't in the docs anywhere.
	size_t work_size = 0;
	int dimensions[] = { sample_count };

	fft_result = hipfftEstimateMany(1, &sample_count, dimensions, 1, sample_count, dimensions, 1, sample_count, HIPFFT_R2C, channel_count, &work_size);
	FFT_RETURN_IF_ERROR(fft_result, "Failed to estimate forward plan.")

	fft_result = hipfftPlanMany(fwd_handle, 1, &sample_count, dimensions, 1, sample_count, dimensions, 1, sample_count, HIPFFT_R2C, channel_count);
	FFT_RETURN_IF_ERROR(fft_result, "Failed to create forward plan.")

	fft_result = hipfftPlanMany(inv_handle, 1, &sample_count, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, channel_count);
	FFT_RETURN_IF_ERROR(fft_result, "Failed to configure inverse plan.")

	return true;
}

__host__ bool 
hilbert::hilbert_transform(hipfftHandle fwd_handle, hipfftHandle inv_handle, float* d_input, hipfftComplex* d_output)
{
	hipfftResult_t fft_result;

	// Exec forward transform
	fft_result = hipfftExecR2C(fwd_handle, d_input, d_output);
	FFT_RETURN_IF_ERROR(fft_result, "Failed to execute forward plan.")

	// Exec reverse transform
	fft_result = hipfftExecC2C(inv_handle, d_output, d_output, HIPFFT_BACKWARD);
	FFT_RETURN_IF_ERROR(fft_result, "Failed to execute inverse plan.")

	return true;
}
