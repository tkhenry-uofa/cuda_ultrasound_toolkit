#include "hip/hip_runtime.h"
#include <iostream>
#include <stdexcept>
#include <chrono>
#include <cuda/std/complex>
#include <hip/hip_math_constants.h>
#include <math_functions.h>

#include <hipcub/hipcub.hpp> 

#include "beamformer.cuh"

__constant__ KernelConstants Constants;

#define MAX_CHANNEL_COUNT 128
#define MAX_TX_COUNT 128

__device__ __inline__  float
beamformer::_kernels::f_num_aprodization(float lateral_dist, float depth, float f_num)
{
	float apro = f_num * lateral_dist / depth/2;
	apro = fminf(apro, 0.5);
	apro = cosf(HIP_PI_F * apro);
	return apro * apro;
}

__device__ hipComplex 
reduce_shared_sum(const hipComplex* sharedVals, const uint channel_count) 
{
	// Each thread will compute a partial sum.
	int tid = threadIdx.x;

	// Compute partial sum over a stripe of the shared memory.
	hipComplex partial_sum = { 0.0f, 0.0f };
	// Each thread processes elements starting at its index and strides by the block size.
	for (int i = tid; i < channel_count; i += blockDim.x) {
		partial_sum = ADD_F2( sharedVals[i], partial_sum);
	}

	__shared__ hipComplex aux[MAX_TX_COUNT];
	aux[tid] = partial_sum;
	__syncthreads();

	// Perform iterative tree-based reduction.
	// Since blockDim.x is 128, we reduce until we have one value.
	for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
		if (tid < stride) {
			aux[tid] = ADD_F2(aux[tid], aux[tid + stride]);
		}
		__syncthreads();
	}
	return aux[0];
}

// Returns true if this element should be used for mixes
__device__ __inline__ bool
offset_mixes(int transmit, int element, int mixes_spacing, int offset, int pivot)
{
	int transmit_offset = 0;
	int element_offset = 0;


	if (transmit >= pivot) element_offset = offset;
	if (element >= pivot) transmit_offset = offset;
	
	if (element % mixes_spacing != element_offset && transmit % mixes_spacing != transmit_offset)
	{
		return false;
	}

	return true;
}

__device__ inline float
beamformer::_kernels::calc_tx_distance(float3 vox_loc, float2* max_lateral_dists)
{
	float tx_distance = 0.0f;
	if (Constants.tx_type == TX_X_FOCUS)
	{
		tx_distance = sqrt(powf(Constants.src_pos.z - vox_loc.z, 2) + powf(Constants.src_pos.x - vox_loc.x, 2)) + Constants.src_pos.z;

		float tx_angle = atan2f(Constants.xdc_maxes.x, -Constants.src_pos.z);

		max_lateral_dists->x = Constants.xdc_maxes.x + vox_loc.z * tanf(tx_angle);
		max_lateral_dists->y = Constants.xdc_maxes.y * 2;

		float2 lateral_ratios = { vox_loc.x / max_lateral_dists->x , vox_loc.y / max_lateral_dists->y };

		if (lateral_ratios.x >= 1.0f || lateral_ratios.y >= 1.0f) return;
		if (lateral_ratios.x <= -1.0f || lateral_ratios.y <= -1.0f) return;

	}
	else if (Constants.tx_type == TX_Y_FOCUS)
	{
		tx_distance = sqrt(powf(Constants.src_pos.z - vox_loc.z, 2) + powf(Constants.src_pos.y - vox_loc.y, 2)) + Constants.src_pos.z;

		float tx_angle = atan2f(Constants.xdc_maxes.y, -Constants.src_pos.z);


		max_lateral_dists->x = Constants.xdc_maxes.x * 2;
		max_lateral_dists->y = Constants.xdc_maxes.y + vox_loc.z * tanf(tx_angle);

		float2 lateral_ratios = { vox_loc.x / max_lateral_dists->x , vox_loc.y / max_lateral_dists->y };

		if (lateral_ratios.x >= 1.0f || lateral_ratios.y >= 1.0f) return;
		if (lateral_ratios.x <= -1.0f || lateral_ratios.y <= -1.0f) return;

	}
	else
	{
		tx_distance = vox_loc.z;
		max_lateral_dists->x = 2 * Constants.xdc_maxes.x;
		max_lateral_dists->y = 2 * Constants.xdc_maxes.y;
	}

	return tx_distance;
}

__global__ void
beamformer::_kernels::double_loop(const hipComplex* rfData, hipComplex* volume, float samples_per_meter, uint64* times)
{
	uint tid = threadIdx.x;
	uint64 start_time;
	if (tid == 0)
	{
		start_time = clock64();
	}

	uint xy_voxel = threadIdx.x + blockIdx.x * blockDim.x;

	if (xy_voxel > Constants.voxel_dims.x * Constants.voxel_dims.y)
	{
		return;
	}

	uint x_voxel = xy_voxel % Constants.voxel_dims.x;
	uint y_voxel = xy_voxel / Constants.voxel_dims.x;
	uint z_voxel = blockIdx.y;

	size_t volume_offset = z_voxel * Constants.voxel_dims.x * Constants.voxel_dims.y + y_voxel * Constants.voxel_dims.x + x_voxel;

	const float3 vox_loc =
	{
		Constants.volume_mins.x + x_voxel * Constants.resolutions.x,
		Constants.volume_mins.y + y_voxel * Constants.resolutions.y,
		Constants.volume_mins.z + z_voxel * Constants.resolutions.z,
	};

	float2 max_lateral_dists;
	float tx_distance = calc_tx_distance(vox_loc, &max_lateral_dists);
	float apro_depth = vox_loc.z / Constants.z_max;
	hipComplex total = {0.0f, 0.0f}, value;
	float incoherent_sum = 0.0f;

	float3 rx_vec = { Constants.xdc_mins.x - vox_loc.x + Constants.pitches.x/2, Constants.xdc_mins.y - vox_loc.y + Constants.pitches.y / 2, vox_loc.z };	
	//float3 rx_vec = { Constants.xdc_mins.x - vox_loc.x + Constants.pitches.x / 2, Constants.xdc_mins.y - vox_loc.y, vox_loc.z };
	//float3 rx_vec = { vox_loc.x - Constants.pitches.x / 2, Constants.xdc_mins.y - vox_loc.y + Constants.pitches.y / 2, vox_loc.z };
	//float3 rx_vec = { vox_loc.x - Constants.pitches.x / 2, vox_loc.y - Constants.pitches.y / 2, vox_loc.z };

	uint delay_samples = 12;


	float starting_x = rx_vec.x;
	float apro;
	size_t channel_offset = 0;
	uint sample_count = Constants.sample_count;
	uint scan_index;
	uint channel_count = Constants.channel_count;

	int mixes_number = 128;
	int mixes_spacing = 128/mixes_number;
	int mixes_offset = 0;
	int total_used_channels = 0;
	//int mixes_offset = mixes_spacing / 2;
	for (int t = 0; t < Constants.tx_count; t++)
	{
		for (int e = 0; e < channel_count; e++)
		{
			if (!offset_mixes(t, e, mixes_spacing, mixes_offset, 64))
			{
				rx_vec.x += Constants.pitches.x;
				continue;
			}

			channel_offset = channel_count * sample_count * t + sample_count * e;

			float2 lateral_ratios = { rx_vec.x / max_lateral_dists.x, rx_vec.y / max_lateral_dists.y };

			scan_index = (uint)((NORM_F3(rx_vec) + tx_distance) * samples_per_meter + 0);
			value = __ldg(&rfData[channel_offset + scan_index - 1]);

			float apro_argument = NORM_F2(lateral_ratios);
			apro = f_num_aprodization(apro_argument, apro_depth, 0.0);
			value = SCALE_F2(value, apro);

			total = ADD_F2(total, value);
			incoherent_sum += NORM_SQUARE_F2(value);

			rx_vec.x += Constants.pitches.x;
			total_used_channels++;

		}
		rx_vec.y += Constants.pitches.y;
		rx_vec.x = starting_x;
	}

	float coherent_sum = NORM_SQUARE_F2(total);

	float coherency_factor = coherent_sum / (incoherent_sum * total_used_channels);
	//volume[volume_offset] = SCALE_F2(total, coherency_factor);;

	volume[volume_offset] = total;

	if (tid == 0)
	{
		uint64 end_time = clock64();
		times[0] = end_time - start_time;
		times[1] = 0;
	}
}

__global__ void
beamformer::_kernels::per_channel_beamform(const hipComplex* rfData, hipComplex* volume, float samples_per_meter, uint readi_group_id)
{
	uint tid = threadIdx.x;

	uint channel_id = threadIdx.x;

	__shared__ hipComplex das_samples[MAX_CHANNEL_COUNT]; // 128

	uint x_voxel = blockIdx.x;
	uint y_voxel = blockIdx.y;
	uint z_voxel = blockIdx.z;

	size_t volume_offset = z_voxel * Constants.voxel_dims.x * Constants.voxel_dims.y + y_voxel * Constants.voxel_dims.x + x_voxel;

	const float3 vox_loc =
	{
		Constants.volume_mins.x + x_voxel * Constants.resolutions.x,
		Constants.volume_mins.y + y_voxel * Constants.resolutions.y,
		Constants.volume_mins.z + z_voxel * Constants.resolutions.z,
	};

	uint transmit_count = Constants.tx_count;

	float2 max_lateral_dists;
	float tx_distance = calc_tx_distance(vox_loc, &max_lateral_dists);
	float apro_depth = vox_loc.z / Constants.z_max;

	float3 rx_vec =	  { Constants.xdc_mins.x - vox_loc.x + channel_id * Constants.pitches.x + Constants.pitches.x / 2, 
						Constants.xdc_mins.y - vox_loc.y + Constants.pitches.y / 2, 
						vox_loc.z };

	uint delay_samples = 12;
	float apro;
	size_t channel_offset = 0;
	uint sample_count = Constants.sample_count;
	uint scan_index;
	uint channel_count = Constants.channel_count;
	uint sample_offset = channel_id;
	hipComplex value;
	hipComplex channel_total = { 0.0f,0.0f };
	for (int t = 0; t < transmit_count; t++)
	{
		channel_offset = channel_count * sample_count * t + sample_count * channel_id;

		float2 lateral_ratios = { rx_vec.x / max_lateral_dists.x, rx_vec.y / max_lateral_dists.y };

		scan_index = (uint)((NORM_F3(rx_vec) + tx_distance) * samples_per_meter + 0);
		value = __ldg(&rfData[channel_offset + scan_index - 1]);

		float apro_argument = NORM_F2(lateral_ratios);
		apro = f_num_aprodization(apro_argument, apro_depth, 0.0);
		value = SCALE_F2(value, apro);

		channel_total = ADD_F2(channel_total,value);

		rx_vec.y += Constants.pitches.y;
	}

	__syncthreads();

	das_samples[channel_id] = channel_total;

	hipComplex vox_total = reduce_shared_sum(das_samples, channel_count);

	if (channel_id == 0)
	{
		volume[volume_offset] = vox_total;
	}

}


bool
beamformer::per_channel_beamform(hipComplex* d_volume, const hipComplex* d_rf_data, float3 focus_pos, float samples_per_meter)
{

	TransmitType transmit_type;

	if (focus_pos.z == 0.0f || focus_pos.z == INFINITY)
	{
		transmit_type = TX_PLANE;
	}
	//else if (Session.channel_offset > 0)
	//{
	//	// TX on columns (x) axis so we have x focusing
	//	transmit_type = TX_X_FOCUS;
	//}
	else
	{
		transmit_type = TX_X_FOCUS;
	}

	VolumeConfiguration vol_config = Session.volume_configuration;

	KernelConstants consts =
	{
		Session.decoded_dims.x,
		Session.decoded_dims.y,
		Session.decoded_dims.z,
		vol_config.voxel_counts,
		vol_config.minimums,
		{vol_config.lateral_resolution, vol_config.lateral_resolution, vol_config.axial_resolution},
		focus_pos,
		transmit_type,
		Session.pitches,
		Session.pulse_delay,
		vol_config.maximums.z,
		Session.xdc_mins,
		Session.xdc_maxes
	};
	CUDA_RETURN_IF_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(Constants), &consts, sizeof(KernelConstants)));

	uint64* times, * d_times;

	times = (uint64*)malloc(2 * sizeof(uint64));
	CUDA_RETURN_IF_ERROR(hipMalloc(&d_times, 2 * sizeof(uint64)));

	uint3 vox_counts = vol_config.voxel_counts;

	dim3 grid_dim = { vox_counts.x, vox_counts.y, vox_counts.z};
	dim3 block_dim = { Session.decoded_dims.y, 1, 1 };

	auto start = std::chrono::high_resolution_clock::now();


	_kernels::per_channel_beamform << < grid_dim, block_dim >> > (d_rf_data, d_volume, samples_per_meter, Session.readi_group);

	CUDA_RETURN_IF_ERROR(hipGetLastError());
	CUDA_RETURN_IF_ERROR(hipDeviceSynchronize());

	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "Kernel duration: " << elapsed.count() << " seconds" << std::endl;

	return true;
}




bool
beamformer::beamform(hipComplex* d_volume, const hipComplex* d_rf_data, float3 focus_pos, float samples_per_meter)
{

	TransmitType transmit_type;

	if (focus_pos.z == 0.0f || focus_pos.z == INFINITY)
	{
		transmit_type = TX_PLANE;
	}
	//else if (Session.channel_offset > 0)
	//{
	//	// TX on columns (x) axis so we have x focusing
	//	transmit_type = TX_X_FOCUS;
	//}
	else
	{
		transmit_type = TX_X_FOCUS;
	}

	VolumeConfiguration vol_config = Session.volume_configuration;

	KernelConstants consts =
	{
		Session.decoded_dims.x,
		Session.decoded_dims.y,
		Session.decoded_dims.z,
		vol_config.voxel_counts,
		vol_config.minimums,
		{vol_config.lateral_resolution, vol_config.lateral_resolution, vol_config.axial_resolution},
		focus_pos,
		transmit_type,
		Session.pitches,
		Session.pulse_delay,
		vol_config.maximums.z,
		Session.xdc_mins,
		Session.xdc_maxes
	};
	CUDA_RETURN_IF_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(Constants), &consts, sizeof(KernelConstants)));

	uint64* times, * d_times;

	times = (uint64*)malloc(2 * sizeof(uint64));
	CUDA_RETURN_IF_ERROR(hipMalloc(&d_times, 2 * sizeof(uint64)));

	uint3 vox_counts = vol_config.voxel_counts;
	uint xy_count = vox_counts.x * vox_counts.y;
	dim3 grid_dim = { (uint)ceilf((float)xy_count / MAX_THREADS_PER_BLOCK), (uint)vox_counts.z, 1 };
	dim3 block_dim = { MAX_THREADS_PER_BLOCK, 1, 1 };

	auto start = std::chrono::high_resolution_clock::now();

	_kernels::double_loop << < grid_dim, block_dim >> > (d_rf_data, d_volume, samples_per_meter, d_times);

	CUDA_RETURN_IF_ERROR(hipGetLastError());
	CUDA_RETURN_IF_ERROR(hipDeviceSynchronize());

	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "Kernel duration: " << elapsed.count() << " seconds" << std::endl;

	return true;
}

