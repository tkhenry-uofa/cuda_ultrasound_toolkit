#include "hip/hip_runtime.h"
#include <iostream>
#include <stdexcept>
#include <chrono>
#include <cuda/std/complex>
#include <hip/hip_math_constants.h>
#include <math_functions.h>

#include <hipcub/hipcub.hpp> 

#include "beamformer.cuh"

__constant__ KernelConstants Constants;

__device__ __inline__  float
beamformer::_kernels::f_num_aprodization(float lateral_dist, float depth, float f_num)
{
	float apro = f_num * lateral_dist / depth;
	apro = fminf(apro, 0.5);
	apro = cosf(HIP_PI_F * apro);
	return apro * apro;
}

__global__ void
beamformer::_kernels::delay_and_sum(const hipComplex* rfData, float* volume, float samples_per_meter, const float2* location_array, uint64* times)
{
	__shared__ hipComplex temp[MAX_THREADS_PER_BLOCK/WARP_SIZE];

	uint e = threadIdx.x;

	// Start timing the reduction portion
    uint64 start_time;
    if (e == 0) {
        start_time = clock64();
    }
	const float3 vox_loc =
	{
		Constants.volume_mins.x + blockIdx.x * Constants.resolutions.x,
		Constants.volume_mins.y + blockIdx.y * Constants.resolutions.y,
		Constants.volume_mins.z + blockIdx.z * Constants.resolutions.z,
	};

	float lateral_dist = sqrtf(vox_loc.x * vox_loc.x + vox_loc.y * vox_loc.y);

	float tx_distance;
	int dist_sign = ((vox_loc.z - Constants.src_pos.z) > 0) ? 1 : -1;
	switch (Constants.tx_type)
	{
	case TX_PLANE:
		tx_distance = vox_loc.z;
		break;

	case TX_Y_FOCUS:
		tx_distance = dist_sign * sqrt(powf(Constants.src_pos.z - vox_loc.z, 2) + powf(Constants.src_pos.y - vox_loc.y, 2)) + Constants.src_pos.z;
		break;

	case TX_X_FOCUS:
		tx_distance = dist_sign * sqrt(powf(Constants.src_pos.z - vox_loc.z, 2) + powf(Constants.src_pos.x - vox_loc.x, 2)) + Constants.src_pos.z;
		break;
	}

	hipComplex total, value;
	float3 rx_vec = { ((float)e - 63.5f) * Constants.element_pitch - vox_loc.x, (-63.5f) * Constants.element_pitch - vox_loc.y, vox_loc.z};

	uint64 loop_start;
	if (e == 0) {
		loop_start = clock64();
	}

	uint delay_samples = (uint)roundf(Constants.pulse_delay * Constants.sample_count);
	for (int t = 0; t < 128; t++)
	{
		uint scan_index = (uint)((NORM_F3(rx_vec) + tx_distance) * samples_per_meter + delay_samples);
		size_t channel_offset = (t * Constants.sample_count * Constants.channel_count) + (e * Constants.sample_count);
		value = __ldg(&rfData[channel_offset + scan_index - 1]);
		f_num_aprodization(lateral_dist, vox_loc.z, 1.5);

		total = ADD_F2(total, value);

		rx_vec.y += Constants.element_pitch;
	}

	__syncthreads();

	uint64 reduce_start_time = clock64();
	/* Each warp sums up their totals using intrinsics and stores the output
	 in that warp's index in temp*/
	for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
	{
		double double_value = __shfl_xor_sync(0xFFFFFFFF, *(double*)&total, offset);
		total = ADD_F2(*(hipComplex*)&double_value, total);
	}
	if (e % WARP_SIZE == 0)
	{
		temp[e / WARP_SIZE] = total;
	}
	__syncthreads();
	
	if (e == 0) 
	{
		total = { 0.0f, 0.0f };
		for (int i = 1; i < MAX_THREADS_PER_BLOCK / WARP_SIZE; i++)
		{
			total = ADD_F2(total, temp[i]);
		}
		volume[blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x] = NORM_F2(total);
		uint64 end_time = clock64();
		times[0] = end_time - start_time;
		times[1] = reduce_start_time - loop_start;
	}
}

__global__ void
beamformer::_kernels::double_loop(const hipComplex* rfData, float* volume, float samples_per_meter, uint64* times)
{
	int tid = threadIdx.x;
	uint64 start_time;
	if (tid == 0)
	{
		start_time = clock64();
	}

	uint xy_voxel = threadIdx.x + blockIdx.x * blockDim.x;

	if (xy_voxel > Constants.voxel_dims.x * Constants.voxel_dims.y)
	{
		return;
	}

	uint x_voxel = xy_voxel % Constants.voxel_dims.x;
	uint y_voxel = xy_voxel / Constants.voxel_dims.x;
	uint z_voxel = blockIdx.y;

	float element_pitch = Constants.element_pitch;

	size_t volume_offset = z_voxel * Constants.voxel_dims.x * Constants.voxel_dims.y + y_voxel * Constants.voxel_dims.x + x_voxel;

	const float3 vox_loc =
	{
		Constants.volume_mins.x + x_voxel * Constants.resolutions.x,
		Constants.volume_mins.y + y_voxel * Constants.resolutions.y,
		Constants.volume_mins.z + z_voxel * Constants.resolutions.z,
	};

	float xdc_edge = 63.5f * element_pitch;

	float apro_argument = 0;
	float tx_distance = 0;
	bool diverging = (Constants.src_pos.z < 0.0f);
	if (diverging)
	{
		tx_distance = sqrt(powf(Constants.src_pos.z - vox_loc.z, 2) + powf(Constants.src_pos.y - vox_loc.y, 2)) + Constants.src_pos.z;
		float tx_angle = atan2f(xdc_edge, -Constants.src_pos.z);
		float max_lateral_dist = xdc_edge + vox_loc.z * tanf(tx_angle);
		float2 lateral_ratios = { vox_loc.y / max_lateral_dist, vox_loc.x / xdc_edge };
		lateral_ratios = { fmaxf(lateral_ratios.x, 1),fmaxf(lateral_ratios.y, 1) };

		apro_argument = NORM_F2(lateral_ratios);
		apro_argument = fmaxf(apro_argument, 1);
	}
	else
	{
		tx_distance = vox_loc.z;
		float max_lateral_dist = sqrtf(xdc_edge * xdc_edge);
		apro_argument = sqrt(vox_loc.x * vox_loc.x + vox_loc.y * vox_loc.y) / max_lateral_dist;
		apro_argument = fmaxf(apro_argument, 1);
	}

	float apro_depth = vox_loc.z / Constants.z_max;

	hipComplex total = {0.0f, 0.0f}, value;
	
	uint delay_samples = (uint)roundf(Constants.pulse_delay * 50e6f);

	float3 rx_vec = { -xdc_edge - vox_loc.x, -xdc_edge - vox_loc.y, vox_loc.z };
	float starting_y = rx_vec.y;
	float apro;
	size_t channel_offset = 0;
	uint sample_count = Constants.sample_count;
	uint scan_index;
	for (int t = 0; t < 128; t++)
	{
		for (int e = 0; e < 128; e++)
		{
			scan_index = (uint)((NORM_F3(rx_vec) + tx_distance) * samples_per_meter + delay_samples);
			
			value = __ldg(&rfData[channel_offset + scan_index - 1]);

			apro = f_num_aprodization(apro_argument, apro_depth, 0.5);

			value = SCALE_F2(value, apro);

			if (t == 0) value = SCALE_F2(value, I_SQRT_128);

			total = ADD_F2(total, value);

			rx_vec.y += element_pitch;
			channel_offset += sample_count;

		}
		rx_vec.x += element_pitch;
		rx_vec.y = starting_y;
	}

	float result = sqrtf(total.x * total.x + total.y * total.y);
	volume[volume_offset] = result;

	if (tid == 0)
	{
		uint64 end_time = clock64();
		times[0] = end_time - start_time;
		times[1] = 0;
	}
}

bool
beamformer::beamform(float* d_volume, const hipComplex* d_rf_data, float3 focus_pos, float samples_per_meter)
{

	TransmitType transmit_type;

	if (focus_pos.z == 0.0f)
	{
		transmit_type = TX_PLANE;
	}
	else if (Session.rx_cols)
	{
		// TX on rows (x) axis so we have x focusing
		transmit_type = TX_X_FOCUS;
	}
	else
	{
		transmit_type = TX_Y_FOCUS;
	}

	VolumeConfiguration vol_config = Session.volume_configuration;


	KernelConstants consts =
	{
		Session.decoded_dims.x,
		Session.decoded_dims.y,
		Session.decoded_dims.z,
		vol_config.voxel_counts,
		vol_config.minimums,
		{vol_config.lateral_resolution, vol_config.lateral_resolution, vol_config.axial_resolution},
		focus_pos,
		transmit_type,
		Session.element_pitch,
		Session.pulse_delay,
		vol_config.maximums.z,
	};
	CUDA_RETURN_IF_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(Constants), &consts, sizeof(KernelConstants)));


	uint64 *times, *d_times;

	times = (uint64*)malloc(2 * sizeof(uint64));
	CUDA_RETURN_IF_ERROR(hipMalloc(&d_times, 2 * sizeof(uint64)));

	uint3 vox_counts = vol_config.voxel_counts;
	uint xy_count = vox_counts.x * vox_counts.y;
	dim3 grid_dim = { (uint)ceilf((float)xy_count / MAX_THREADS_PER_BLOCK), (uint)vox_counts.z, 1 };

	dim3 block_dim = { MAX_THREADS_PER_BLOCK, 1, 1 };

	auto start = std::chrono::high_resolution_clock::now();

	_kernels::double_loop << < grid_dim, block_dim >> > (d_rf_data, d_volume, samples_per_meter, d_times);
	

	CUDA_RETURN_IF_ERROR(hipGetLastError());
	CUDA_RETURN_IF_ERROR(hipDeviceSynchronize());

	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "Kernel duration: " << elapsed.count() << " seconds" << std::endl;

	std::cout << "First volume value: " << sample_value(d_volume) << std::endl;

	return true;

}

