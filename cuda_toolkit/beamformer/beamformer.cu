#include "hip/hip_runtime.h"
#include <iostream>
#include <stdexcept>
#include <chrono>
#include <cuda/std/complex>
#include <hip/hip_math_constants.h>
#include <math_functions.h>

#include <hipcub/hipcub.hpp> 

#include "beamformer.cuh"

__constant__ KernelConstants Constants;

#define PULSE_DELAY 0

__device__ __inline__ float
old_beamformer::_kernels::f_num_aprodization(float lateral_dist, float depth, float f_num)
{
	float apro = f_num * lateral_dist / depth;
	apro = fminf(apro, 0.5);
	apro = cosf(HIP_PI_F * apro);
	return apro * apro;
}

__global__ void
old_beamformer::_kernels::delay_and_sum(const hipComplex* rfData, float* volume, float samples_per_meter, const float2* location_array, uint64* times)
{
	__shared__ hipComplex temp[MAX_THREADS_PER_BLOCK/WARP_SIZE];

	uint e = threadIdx.x;

	// Start timing the reduction portion
    uint64 start_time;
    if (e == 0) {
        start_time = clock64();
    }
	const float3 vox_loc =
	{
		Constants.volume_mins.x + blockIdx.x * Constants.resolutions.x,
		Constants.volume_mins.y + blockIdx.y * Constants.resolutions.y,
		Constants.volume_mins.z + blockIdx.z * Constants.resolutions.z,
	};

	float lateral_dist = sqrtf(vox_loc.x * vox_loc.x + vox_loc.y * vox_loc.y);

	float tx_distance;
	int dist_sign = ((vox_loc.z - Constants.src_pos.z) > 0) ? 1 : -1;
	switch (Constants.tx_type)
	{
	case TX_PLANE:
		tx_distance = vox_loc.z;
		break;

	case TX_Y_FOCUS:
		tx_distance = dist_sign * sqrt(powf(Constants.src_pos.z - vox_loc.z, 2) + powf(Constants.src_pos.y - vox_loc.y, 2)) + Constants.src_pos.z;
		break;

	case TX_X_FOCUS:
		tx_distance = dist_sign * sqrt(powf(Constants.src_pos.z - vox_loc.z, 2) + powf(Constants.src_pos.x - vox_loc.x, 2)) + Constants.src_pos.z;
		break;
	}

	hipComplex total, value;
	float3 rx_vec = { ((float)e - 63.5f) * Constants.element_pitch - vox_loc.x, (-63.5f) * Constants.element_pitch - vox_loc.y, vox_loc.z};

	uint64 loop_start;
	if (e == 0) {
		loop_start = clock64();
	}

	for (int t = 0; t < 128; t++)
	{
		uint scan_index = (uint)((NORM_F3(rx_vec) + tx_distance) * samples_per_meter + PULSE_DELAY);
		size_t channel_offset = (t * Constants.sample_count * Constants.channel_count) + (e * Constants.sample_count);
		value = __ldg(&rfData[channel_offset + scan_index - 1]);
		f_num_aprodization(lateral_dist, vox_loc.z, 1.5);

		total = ADD_F2(total, value);

		rx_vec.y += Constants.element_pitch;
	}

	__syncthreads();

	uint64 reduce_start_time = clock64();
	/* Each warp sums up their totals using intrinsics and stores the output
	 in that warp's index in temp*/
	for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
	{
		double double_value = __shfl_xor_sync(0xFFFFFFFF, *(double*)&total, offset);
		total = ADD_F2(*(hipComplex*)&double_value, total);
	}
	if (e % WARP_SIZE == 0)
	{
		temp[e / WARP_SIZE] = total;
	}
	__syncthreads();
	
	if (e == 0) 
	{
		total = { 0.0f, 0.0f };
		for (int i = 1; i < MAX_THREADS_PER_BLOCK / WARP_SIZE; i++)
		{
			total = ADD_F2(total, temp[i]);
		}
		volume[blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x] = NORM_F2(total);
		uint64 end_time = clock64();
		times[0] = end_time - start_time;
		times[1] = reduce_start_time - loop_start;
	}
}

__global__ void
old_beamformer::_kernels::double_loop(const hipComplex* rfData, float* volume, float samples_per_meter, uint64* times)
{
	int tid = threadIdx.x;
	uint64 start_time;
	if (tid == 0)
	{
		start_time = clock64();
	}

	uint xy_voxel = threadIdx.x + blockIdx.x * blockDim.x;

	if (xy_voxel > Constants.voxel_dims.x * Constants.voxel_dims.y)
	{
		return;
	}

	uint x_voxel = xy_voxel % Constants.voxel_dims.x;
	uint y_voxel = xy_voxel / Constants.voxel_dims.x;
	uint z_voxel = blockIdx.y;

	float element_pitch = Constants.element_pitch;

	size_t volume_offset = z_voxel * Constants.voxel_dims.x * Constants.voxel_dims.y + y_voxel * Constants.voxel_dims.x + x_voxel;

	const float3 vox_loc =
	{
		Constants.volume_mins.x + x_voxel * Constants.resolutions.x,
		Constants.volume_mins.y + y_voxel * Constants.resolutions.y,
		Constants.volume_mins.z + z_voxel * Constants.resolutions.z,
	};

	float lateral_dist = sqrtf(vox_loc.x * vox_loc.x + vox_loc.y * vox_loc.y);

	float tx_distance = vox_loc.z;

	hipComplex total = {0.0f, 0.0f}, value;
	
	float3 rx_vec = { (- 63.5f) * element_pitch - vox_loc.x, ( - 63.5f) * element_pitch - vox_loc.y, vox_loc.z };
	float starting_y = rx_vec.y;
	float apro;
	size_t channel_offset = 0;
	uint sample_count = Constants.sample_count;
	for (int t = 0; t < 128; t++)
	{
		for (int e = 0; e < 128; e++)
		{
			uint scan_index = (uint)((NORM_F3(rx_vec) + tx_distance) * samples_per_meter + PULSE_DELAY);
			
			value = __ldg(&rfData[channel_offset + scan_index - 1]);

			/*apro = f_num_aprodization(lateral_dist, vox_loc.z, 0.5);

			value = SCALE_F2(value, apro);*/
			total = ADD_F2(total, value);

			rx_vec.y += element_pitch;
			channel_offset += sample_count;

		}
		rx_vec.x += element_pitch;
		rx_vec.y = starting_y;
	}
	volume[volume_offset] = NORM_F2(total);

	if (tid == 0)
	{
		uint64 end_time = clock64();
		times[0] = end_time - start_time;
		times[1] = 0;
	}
}

bool
old_beamformer::beamform(float* d_volume, const hipComplex* d_rf_data, const float2* d_loc_data, float3 src_pos, float samples_per_meter)
{

	TransmitType transmit_type;

	if (src_pos.z == 0.0f)
	{
		transmit_type = TX_PLANE;
	}
	else if (Session.rx_cols)
	{
		// TX on rows (x) axis so we have x focusing
		transmit_type = TX_X_FOCUS;
	}
	else
	{
		transmit_type = TX_Y_FOCUS;
	}

	VolumeConfiguration vol_config = Session.volume_configuration;


	KernelConstants consts =
	{
		Session.decoded_dims.x,
		Session.decoded_dims.y,
		Session.decoded_dims.z,
		vol_config.voxel_counts,
		vol_config.minimums,
		{vol_config.lateral_resolution, vol_config.lateral_resolution, vol_config.axial_resolution},
		src_pos,
		transmit_type,
		Session.element_pitch,
	};
	CUDA_RETURN_IF_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(Constants), &consts, sizeof(KernelConstants)));


	uint64 *times, *d_times;

	times = (uint64*)malloc(2 * sizeof(uint64));
	CUDA_RETURN_IF_ERROR(hipMalloc(&d_times, 2 * sizeof(uint64)));

	uint3 vox_counts = vol_config.voxel_counts;
	uint xy_count = vox_counts.x * vox_counts.y;
	dim3 grid_dim = { (uint)ceilf((float)xy_count / MAX_THREADS_PER_BLOCK), (uint)vox_counts.z, 1 };

	dim3 block_dim = { MAX_THREADS_PER_BLOCK, 1, 1 };

	auto start = std::chrono::high_resolution_clock::now();

	_kernels::double_loop << < grid_dim, block_dim >> > (d_rf_data, d_volume, samples_per_meter, d_times);
	

	CUDA_RETURN_IF_ERROR(hipMemcpy(times, d_times, 2 * sizeof(uint64), hipMemcpyDefault));
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	float clockRate = prop.clockRate * 1e3;  // Convert kHz to Hz

	float total_time = (float)times[0] / clockRate;
	float reduce_time = (float)times[1] / clockRate;

	//std::cout << "Loop kernel time: " << total_time << std::endl;
	//std::cout << "Reduction time: " << reduce_time << std::endl;

	grid_dim = { vox_counts.x, vox_counts.y, vox_counts.z };

	//_kernels::delay_and_sum << < grid_dim, block_dim >> > (d_rf_data, d_volume, samples_per_meter, d_loc_data, d_times);
	//CUDA_RETURN_IF_ERROR(hipMemcpy(times, d_times, 2 * sizeof(uint64), hipMemcpyDefault));

	//total_time = (float)times[0] / clockRate;
	//reduce_time = (float)times[1] / clockRate;
	//
	//std::cout << "Unrolled time: " << total_time << std::endl;

	CUDA_RETURN_IF_ERROR(hipGetLastError());
	CUDA_RETURN_IF_ERROR(hipDeviceSynchronize());

	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "Kernel duration: " << elapsed.count() << " seconds" << std::endl;

	return true;

}

