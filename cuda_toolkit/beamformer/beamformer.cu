#include "hip/hip_runtime.h"
#include <iostream>
#include <stdexcept>
#include <chrono>
#include <cuda/std/complex>
#include <hip/hip_math_constants.h>
#include <math_functions.h>

#include <hipcub/hipcub.hpp> 

#include "beamformer.cuh"

__constant__ KernelConstants Constants;

#define MAX_CHANNEL_COUNT 128
#define MAX_TX_COUNT 128

__device__ __inline__  float
beamformer::_kernels::f_num_apodization(float lateral_dist, float depth, float f_num)
{
	// When lateral_dist > depth / f_num clamp the argument to pi/2 so that the cos is 0
	// Otherwise the ratio will map between 0 and pi/2 forming a hann window
	float apo = f_num * (lateral_dist / depth) /2;
	apo = fminf(apo, 0.5);
	apo = cosf(HIP_PI_F * apo);
	return apo * apo; // cos^2
}

__device__ hipComplex 
reduce_shared_sum(const hipComplex* sharedVals, const uint channel_count) 
{
	// Each thread will compute a partial sum.
	int tid = threadIdx.x;

	// Compute partial sum over a stripe of the shared memory.
	hipComplex partial_sum = { 0.0f, 0.0f };
	// Each thread processes elements starting at its index and strides by the block size.
	for (int i = tid; i < channel_count; i += blockDim.x) {
		partial_sum = ADD_F2( sharedVals[i], partial_sum);
	}

	__shared__ hipComplex aux[MAX_TX_COUNT];
	aux[tid] = partial_sum;
	__syncthreads();

	// Perform iterative tree-based reduction.
	// Since blockDim.x is 128, we reduce until we have one value.
	for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
		if (tid < stride) {
			aux[tid] = ADD_F2(aux[tid], aux[tid + stride]);
		}
		__syncthreads();
	}
	return aux[0];
}

// Returns true if this element should be used for mixes
__device__ __inline__ bool
offset_mixes(int transmit, int element, int mixes_spacing, int offset, int pivot)
{
	int transmit_offset = 0;
	int element_offset = 0;


	if (transmit >= pivot) element_offset = offset;
	if (element >= pivot) transmit_offset = offset;
	
	if (element % mixes_spacing != element_offset && transmit % mixes_spacing != transmit_offset)
	{
		return false;
	}

	return true;
}

__device__ inline bool
beamformer::_kernels::check_ranges(float3 vox_loc, float f_number, float2 array_edges)
{
	// Get the max aperture size for this depth
	float lateral_extent = vox_loc.z / f_number;

	// Model 2 1D apertures to maintain square planes
	float x_extent = lateral_extent + array_edges.x;
	float y_extent = lateral_extent + array_edges.y;

	return (abs(vox_loc.x) < x_extent && abs(vox_loc.y) < y_extent);
}

__device__ inline float3
beamformer::_kernels::calc_tx_distance(float3 vox_loc, float3 source_pos)
{
	float3 tx_distance;
	if (Constants.tx_type == TX_X_FOCUS)
	{
		tx_distance = { source_pos.x - vox_loc.x, 0.0f, source_pos.z - vox_loc.z};
	}
	else if (Constants.tx_type == TX_Y_FOCUS)
	{
		tx_distance = { 0.0f, source_pos.y - vox_loc.y, source_pos.z - vox_loc.z };
	}
	else
	{
		tx_distance = {0.0f, 0.0f, vox_loc.z };
	}

	return tx_distance;
}

__device__ inline float
beamformer::_kernels::total_path_length(float3 tx_vec, float3 rx_vec, float focal_depth, float vox_depth)
{
	// If the voxel is shallower than the focus we need to subtract the tx vec
	int sign = vox_depth > focal_depth ? 1 : -1;
	return focal_depth + NORM_F3(rx_vec) + NORM_F3(tx_vec) * sign;
}

__global__ void
beamformer::_kernels::per_voxel_beamform(const hipComplex* rfData, hipComplex* volume, uint readi_group_id, float* hadamard)
{
	uint xy_voxel = threadIdx.x + blockIdx.x * blockDim.x;

	if (xy_voxel > Constants.voxel_dims.x * Constants.voxel_dims.y)
	{
		return;
	}

	uint x_voxel = xy_voxel % Constants.voxel_dims.x;
	uint y_voxel = xy_voxel / Constants.voxel_dims.x;
	uint z_voxel = blockIdx.y;

	size_t volume_offset = z_voxel * Constants.voxel_dims.x * Constants.voxel_dims.y + y_voxel * Constants.voxel_dims.x + x_voxel;

	const float3 vox_loc =
	{
		Constants.volume_mins.x + x_voxel * Constants.resolutions.x,
		Constants.volume_mins.y + y_voxel * Constants.resolutions.y,
		Constants.volume_mins.z + z_voxel * Constants.resolutions.z,
	};

	// If the voxel is out of the f_number defined range for all elements skip it
	if (!check_ranges(vox_loc, Constants.f_number, Constants.xdc_maxes)) return;
	
	float3 src_pos = Constants.src_pos;
	if (Constants.sequence == DAS_ID_FORCES)
	{
		src_pos.x = Constants.xdc_mins.x + Constants.pitches.x / 2;
		src_pos.z = 0; // Ignoring the elevational focus as it is out of plane
	}


	float3 tx_vec = calc_tx_distance(vox_loc, src_pos);

	float3 rx_vec = { Constants.xdc_mins.x - vox_loc.x + Constants.pitches.x / 2, Constants.xdc_mins.y - vox_loc.y + Constants.pitches.y / 2, vox_loc.z };
	
	if (Constants.sequence == DAS_ID_FORCES)
	{
		rx_vec.y = 0;
	}

	int readi_group_size = Constants.channel_count / Constants.tx_count;
	uint hadamard_offset = Constants.channel_count * readi_group_id;
	uint delay_samples = 12;

	hipComplex total = { 0.0f, 0.0f }, value;
	float incoherent_sum = 0.0f;

	float starting_x = rx_vec.x;
	float apo;
	size_t channel_offset = 0;
	uint sample_count = Constants.sample_count;
	uint scan_index;
	uint channel_count = Constants.channel_count;
	float samples_per_meter = Constants.samples_per_meter;

	int mixes_number = 128;
	int mixes_spacing = 128/mixes_number;
	int mixes_offset = 0;
	int total_used_channels = 0;
	//int mixes_offset = mixes_spacing / 2;

	float total_distance = 0.0f;
	for (int t = 0; t < Constants.tx_count; t++)
	{
		for (int e = 0; e < channel_count; e++)
		{
			channel_offset = channel_count * sample_count * t + sample_count * e;
			for (int g = 0; g < readi_group_size; g++)
			{
				if (!offset_mixes(t, e, mixes_spacing, mixes_offset, 64))
				{
					rx_vec.x += Constants.pitches.x;
					continue;
				}

				total_distance = total_path_length(tx_vec, rx_vec, src_pos.z, vox_loc.z);
				scan_index = (uint)(total_distance * samples_per_meter + delay_samples);
				value = __ldg(&rfData[channel_offset + scan_index - 1]);


				apo = f_num_apodization(NORM_F2(rx_vec), vox_loc.z, Constants.f_number);
				value = SCALE_F2(value, apo);

				// This acts as the final decoding step for the data within the readi group
				// If readi is turned off this will just scan the first row of the hadamard matrix (all 1s)
				value = SCALE_F2(value, hadamard[hadamard_offset + g]);

				total = ADD_F2(total, value);
				incoherent_sum += NORM_SQUARE_F2(value);

				rx_vec.x += Constants.pitches.x;
				total_used_channels++;

			}
		}
		rx_vec.x = starting_x;

		if (Constants.sequence == TransmitModes::DAS_ID_HERCULES)
		{
			rx_vec.y += Constants.pitches.x;
		}
		else if (Constants.sequence == TransmitModes::DAS_ID_FORCES)
		{
			tx_vec.x += Constants.pitches.x;
		}

		
	}

	float coherent_sum = NORM_SQUARE_F2(total);

	//float coherency_factor = coherent_sum / (incoherent_sum * total_used_channels);
	//volume[volume_offset] = SCALE_F2(total, coherency_factor);

	volume[volume_offset] = total;
}

__global__ void
beamformer::_kernels::per_channel_beamform(const hipComplex* rfData, hipComplex* volume, uint readi_group_id, float* hadamard)
{
	uint tid = threadIdx.x;

	uint channel_id = threadIdx.x;

	__shared__ hipComplex das_samples[MAX_CHANNEL_COUNT * 2]; // 128 * 2

	uint x_voxel = blockIdx.x;
	uint y_voxel = blockIdx.y;
	uint z_voxel = blockIdx.z;

	size_t volume_offset = z_voxel * Constants.voxel_dims.x * Constants.voxel_dims.y + y_voxel * Constants.voxel_dims.x + x_voxel;

	const float3 vox_loc =
	{
		Constants.volume_mins.x + x_voxel * Constants.resolutions.x,
		Constants.volume_mins.y + y_voxel * Constants.resolutions.y,
		Constants.volume_mins.z + z_voxel * Constants.resolutions.z,
	};

	uint transmit_count = Constants.tx_count;

	// If the voxel is out of the f_number defined range for all elements skip it
	if (!check_ranges(vox_loc, Constants.f_number, Constants.xdc_maxes)) return;

	float3 tx_vec = calc_tx_distance(vox_loc, Constants.src_pos);

	float3 rx_vec =	  { Constants.xdc_mins.x - vox_loc.x + channel_id * Constants.pitches.x + Constants.pitches.x / 2, 
						Constants.xdc_mins.y - vox_loc.y + Constants.pitches.y / 2, 
						vox_loc.z };



	uint readi_group_size = Constants.channel_count / Constants.tx_count;
	uint hadamard_offset = Constants.channel_count * readi_group_id;
	float f_number = Constants.f_number;
	uint delay_samples = 12;
	float apo;
	size_t channel_offset = 0;
	uint sample_count = Constants.sample_count;
	uint scan_index;
	uint channel_count = Constants.channel_count;
	hipComplex value;
	hipComplex channel_total = { 0.0f,0.0f };
	float incoherent_total = 0.0f;
	float total_distance = 0.0f;
	float samples_per_meter = Constants.samples_per_meter;
	for (int t = 0; t < transmit_count; t++)
	{
		channel_offset = channel_count * sample_count * t + sample_count * channel_id;

		for (int g = 0; g < readi_group_size; g++)
		{
			total_distance = total_path_length(tx_vec, rx_vec, Constants.src_pos.z, vox_loc.z);
			scan_index = (uint)(total_distance * samples_per_meter + delay_samples);
			value = __ldg(&rfData[channel_offset + scan_index - 1]);

			apo = f_num_apodization(NORM_F2(rx_vec), vox_loc.z, f_number);

			// This acts as the final decoding step for the data within the readi group
			// If readi is turned off this will just scan the first row of the hadamard matrix (all 1s)
			value = SCALE_F2(value, hadamard[hadamard_offset + g]);

			value = SCALE_F2(value, apo);

			incoherent_total += NORM_SQUARE_F2(value);
			channel_total = ADD_F2(channel_total, value);

			rx_vec.y += Constants.pitches.x;
		}

	}

	__syncthreads();

	das_samples[channel_id] = channel_total;
	das_samples[channel_id + MAX_CHANNEL_COUNT] = { incoherent_total, 0.0f };

	hipComplex vox_total = reduce_shared_sum(das_samples, channel_count);

	__syncthreads();
	//hipComplex incoherent_sum = reduce_shared_sum(das_samples + MAX_CHANNEL_COUNT, channel_count);

	__syncthreads();
	if (channel_id == 0)
	{
		//float coherence_factor = NORM_SQUARE_F2(vox_total) / (incoherent_sum.x * channel_count);
		volume[volume_offset] = vox_total;
		//volume[volume_offset] = SCALE_F2(vox_total, coherence_factor);
	}

}


bool
beamformer::beamform(hipComplex* d_volume, const hipComplex* d_rf_data, float3 focus_pos, float samples_per_meter, float f_number)
{

	TransmitType transmit_type;


	if (focus_pos.z == INFINITY)
	{
		focus_pos.z = 0.0f; // This lets us reuse focusing code for plane waves
		transmit_type = TX_PLANE;
	}
	//else if (Session.channel_offset > 0)
	//{
	//	// TX on columns (x) axis so we have x focusing
	//	transmit_type = TX_X_FOCUS;
	//}
	else
	{
		transmit_type = TX_X_FOCUS;
	}

	VolumeConfiguration vol_config = Session.volume_configuration;

	KernelConstants consts =
	{
		Session.decoded_dims.x,
		Session.decoded_dims.y,
		Session.decoded_dims.z,
		vol_config.voxel_counts,
		vol_config.minimums,
		{vol_config.lateral_resolution, vol_config.lateral_resolution, vol_config.axial_resolution},
		focus_pos,
		transmit_type,
		Session.pitches,
		Session.pulse_delay,
		vol_config.maximums.z,
		Session.xdc_mins,
		Session.xdc_maxes,
		f_number,
		samples_per_meter,
		Session.sequence,
	};
	CUDA_RETURN_IF_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(Constants), &consts, sizeof(KernelConstants)));

	uint3 vox_counts = vol_config.voxel_counts;

	bool per_voxel = true;
	auto start = std::chrono::high_resolution_clock::now();

	if (per_voxel)
	{
		uint xy_count = vox_counts.x * vox_counts.y;
		dim3 grid_dim = { (uint)ceilf((float)xy_count / MAX_THREADS_PER_BLOCK), (uint)vox_counts.z, 1 };
		dim3 block_dim = { MAX_THREADS_PER_BLOCK, 1, 1 };

		_kernels::per_voxel_beamform << < grid_dim, block_dim >> > (d_rf_data, d_volume, Session.readi_group, Session.d_hadamard);
	}
	else
	{
		dim3 grid_dim = { vox_counts.x, vox_counts.y, vox_counts.z };
		dim3 block_dim = { Session.decoded_dims.y, 1, 1 };

		_kernels::per_channel_beamform << < grid_dim, block_dim >> > (d_rf_data, d_volume, Session.readi_group, Session.d_hadamard);
	}


	CUDA_RETURN_IF_ERROR(hipGetLastError());
	CUDA_RETURN_IF_ERROR(hipDeviceSynchronize());

	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "Kernel duration: " << elapsed.count() << " seconds" << std::endl;

	return true;
}

