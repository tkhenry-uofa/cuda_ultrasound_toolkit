#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>

#include <chrono>

#include "hadamard.cuh"

__global__ void
hadamard::_kernels::generate_hadamard(float* hadamard, int prev_size, int final_size)
{
	int row = threadIdx.x + blockDim.x * blockIdx.x;
	int col = threadIdx.y + blockDim.y * blockIdx.y;

	bool top = row < prev_size;
	bool left = col < prev_size;

	// Index to get the value from the previous iteration
	int prev_row = top ? row : row - prev_size;
	int prev_col = left ? col : col - prev_size;

	if (!top && !left)
	{
		// If we are bottom right make a negative copy of top left
		hadamard[row * final_size + col] = -1.0 * hadamard[prev_row * final_size + prev_col];
	}
	else if (top != left)
	{
		// If we are top right or bottom left copy the top left value
		hadamard[row * final_size + col] = hadamard[prev_row * final_size + prev_col];
	}
}

__host__ void 
hadamard::_host::print_array(float* out_array, uint size)
{
	std::cout << "Output" << std::endl;
	for (uint i = 0; i < size; i++)
	{
		for (uint j = 0; j < size; j++)
		{
			std::cout << out_array[i * size + j] << " ";
		}

		std::cout << std::endl;
	}

	std::cout << std::endl;
}

__global__ void
hadamard::_kernels::init_hadamard_matrix(float* matrix, int size)
{
	int row = threadIdx.x * blockIdx.x;
	int col = threadIdx.y * blockIdx.y;

	if (row == 0 && col == 0)
	{
		matrix[0] = 1.0f;
	}
	else if (row < size && col < size)
	{
		matrix[row * size + col] = 0.0f;
	}
}

__host__ bool
hadamard::generate_hadamard(uint size, float** dev_ptr)
{
	if (!(ISPOWEROF2(size)))
	{
		return false;
	}

	CUDA_NULL_FREE(*dev_ptr);

	size_t matrix_size = size * size * sizeof(float);
	CUDA_RETURN_IF_ERROR(hipMalloc((void**)dev_ptr, matrix_size));

	uint grid_length;
	dim3 block_dim, grid_dim;

	if (size <= MAX_2D_BLOCK_DIM)
	{
		grid_length = 1;
		block_dim = { size, size, 1 };
	}
	else
	{
		grid_length = (uint)ceil((double)size / MAX_2D_BLOCK_DIM);
		block_dim = { MAX_2D_BLOCK_DIM, MAX_2D_BLOCK_DIM, 1 };
	}

	grid_dim = { grid_length, grid_length, 1 };

	_kernels::init_hadamard_matrix << <grid_dim, block_dim >> > (*dev_ptr, size);


	CUDA_RETURN_IF_ERROR(hipGetLastError());
	CUDA_RETURN_IF_ERROR(hipDeviceSynchronize());

	for (uint i = 2; i <= size; i *= 2)
	{
		if (i <= MAX_2D_BLOCK_DIM)
		{
			block_dim = { i, i, 1 };
			grid_dim = { 1, 1, 1 };
		}
		else
		{
			grid_length = (uint)ceil((double)i / MAX_2D_BLOCK_DIM);

			block_dim = { MAX_2D_BLOCK_DIM, MAX_2D_BLOCK_DIM, 1 };
			grid_dim = { grid_length, grid_length, 1 };
		}

		_kernels::generate_hadamard << <grid_dim, block_dim >> > (*dev_ptr, i / 2, size);

		CUDA_RETURN_IF_ERROR(hipGetLastError());
		CUDA_RETURN_IF_ERROR(hipDeviceSynchronize());
	}

	return true;
}

__host__ bool 
hadamard::hadamard_decode(const float* d_input, float* d_output)
{

	if (!Session.hadamard_generated)
	{
		std::cerr << "Hadamard: Attempted to decode without a valid hadamard matrix" << std::endl;
		return false;
	}

	uint3 dims = Session.decoded_dims;
	uint tx_size = dims.x * dims.y;

	float alpha = 1.0f;
	//float alpha = 1/((float)dims.x/2);
	float beta = 0.0f;

	CUBLAS_THROW_IF_ERR(hipblasSgemm(Session.cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, tx_size, dims.z, dims.z, &alpha, d_input, tx_size, Session.d_hadamard, dims.z, &beta, d_output, tx_size));

	return true;
}

__host__ bool
hadamard::readi_decode(const float* d_input, float* d_output, int group_number)
{

	int row_count = 128;
	int group_size = 8;
	int group_count = 16;

	float* hadamard_slice;
	hipMalloc(&hadamard_slice, row_count * group_size * sizeof(float));

	int hadamard_offset = group_number * group_size; 

	hipMemcpy(hadamard_slice, d_input + hadamard_offset, group_size * sizeof(float), hipMemcpyDeviceToDevice);

	uint3 dims = Session.decoded_dims;
	uint tx_size = dims.x * dims.y;

	float alpha = 1.0f;
	//float alpha = 1/((float)dims.x/2);
	float beta = 0.0f;

	CUBLAS_THROW_IF_ERR(hipblasSgemm(Session.cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, tx_size, row_count, group_size, &alpha, d_input, tx_size, Session.d_hadamard, dims.z, &beta, d_output, tx_size));


	return true;
}








