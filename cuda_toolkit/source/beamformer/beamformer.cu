#include "hip/hip_runtime.h"
#include <chrono>

#include "beamformer_kernels.cuh"
#include "beamformer.h"

namespace beamform
{
    kernels::BeamformerConstants
    params_to_constants(const CudaBeamformerParameters& bp)
    {
        kernels::BeamformerConstants constants;

        constants.sample_count = bp.dec_data_dim[0];
        constants.channel_count = bp.dec_data_dim[1];
        constants.tx_count = bp.dec_data_dim[2];

        constants.xdc_mins = {-bp.xdc_transform[12], -bp.xdc_transform[13]};
        constants.xdc_maxes = {bp.xdc_transform[12], bp.xdc_transform[13]};

        constants.samples_per_meter = bp.sampling_frequency / bp.speed_of_sound;

        constants.pitches = {bp.xdc_element_pitch[0], bp.xdc_element_pitch[1]};
        constants.delay_samples = (int)(bp.time_offset * bp.sampling_frequency);
        constants.sequence = bp.das_shader_id;

        constants.voxel_dims = {bp.output_points[0], bp.output_points[1], bp.output_points[2]};
        constants.volume_mins = {bp.output_min_coordinate[0], bp.output_min_coordinate[1], bp.output_min_coordinate[2]};

        float lateral_resolution = (bp.output_max_coordinate[0] - bp.output_min_coordinate[0])
                                   / (bp.output_points[0] - 1);

        float elevation_resolution = lateral_resolution;
        float axial_resolution = (bp.output_max_coordinate[2] - bp.output_min_coordinate[2])
                                   / (bp.output_points[2] - 1);

        constants.resolutions = {lateral_resolution, elevation_resolution, axial_resolution};
        constants.f_number = bp.f_number;

        constants.mixes_count = bp.mixes_count;
        constants.mixes_offset = bp.mixes_offset;

        constants.readi_group_count = bp.readi_group_count;
        constants.readi_group_id = bp.readi_group_id;

        float3 focal_point = {0.0f, 0.0f, bp.focal_depths[0]};
        constants.focal_point = focal_point;
        if(focal_point.z == INFINITY)
        {
            constants.focal_direction = kernels::FocalDirection::PLANE;
        }
        else if(bp.das_shader_id == SequenceId::HERCULES 
            || bp.das_shader_id == SequenceId::UHURCULES
            || bp.das_shader_id == SequenceId::EPIC_UHERCULES)
        {
            constants.focal_direction = kernels::FocalDirection::YZ_PLANE;
        }
        else
        {
            constants.focal_direction = kernels::FocalDirection::XZ_PLANE;
        }


        return constants;
    }

    bool
    Beamformer::per_voxel_beamform(hipComplex* d_input,
                         hipComplex* d_volume,
                         const CudaBeamformerParameters& bp,
                         const float* d_hadamard)
    {
        auto constants = params_to_constants(bp);
		kernels::copy_kernel_constants(constants);

        std::cout << "Starting beamform." << std::endl;

        if(constants.readi_group_count > 1)
        {
            // We just want the relevant row for this group
            d_hadamard += constants.readi_group_id * constants.readi_group_count;
        }


        uint3 vox_counts = constants.voxel_dims;
        uint xy_count = vox_counts.x * vox_counts.y;
		dim3 grid_dim = { (xy_count + MAX_THREADS_PER_BLOCK -1) / MAX_THREADS_PER_BLOCK, vox_counts.z, 1 };
		dim3 block_dim = { MAX_THREADS_PER_BLOCK, 1, 1 };

        auto start = std::chrono::high_resolution_clock::now();
		kernels::per_voxel_beamform << < grid_dim, block_dim >> > (d_input, d_volume, d_hadamard);

        CUDA_RETURN_IF_ERROR(hipGetLastError());
	    CUDA_RETURN_IF_ERROR(hipDeviceSynchronize());

        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;
        std::cout << "Kernel duration: " << elapsed.count() << " seconds" << std::endl;

        return true;
    }
}